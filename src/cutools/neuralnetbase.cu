// STD Lib Headers
#include <iostream>
#include <vector>
#include <sstream>
#include <string>

// CUDA Headers
#include <hip/hip_runtime.h>
#include <hipDNN.h>

#include "cudaerrorhandling.cuh"
#include "cudadevicestools.cuh"

#include "neuralnetbase.cuh"

void cuNeuralnetbase::setupCudaDevice()
{
    cudaErrorHandler(hipGetDeviceCount(&numdevice));
    devprops.resize(numdevice);

    /*GET CUDA DEVICE PROPERTIES*/
    for (int i=0;i<int(devprops.size());++i)
        cudaErrorHandler(hipGetDeviceProperties(&devprops[i],i));

    printDevProps(devprops);

    cudaErrorHandler(hipSetDevice(1));
};
