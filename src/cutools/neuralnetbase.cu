// STD Lib Headers
#include <iostream>
#include <vector>
#include <sstream>
#include <string>

// CUDA Headers
#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include <hipblas.h>

#include "cudaerrorhandling.cuh"
#include "cudadevicestools.cuh"

#include "neuralnetbase.cuh"

void cuNeuralNetworkbase::m_setupCudaDevice()
{
    cudaErrorHandler(hipGetDeviceCount(&numdevice));
    devprops.resize(numdevice);

    /*GET CUDA DEVICE PROPERTIES*/
    for (int i=0; i<int(devprops.size()); ++i)
        cudaErrorHandler(hipGetDeviceProperties(&devprops[i],i));

    printDevProps(devprops);

    cudaErrorHandler(hipSetDevice(0));
};

void cuNeuralNetworkbase::m_createHandles()
{
    std::cout << "Creating cuDNN Handles!" << "\n";
    cudnnErrorHandler(hipdnnCreate(&cudnnHandle));
    cudnnErrorHandler(hipdnnCreateTensorDescriptor(&srcTensorDesc));
    cudnnErrorHandler(hipdnnCreateTensorDescriptor(&dstTensorDesc));
    std::cout << " Running cuDNN version: " << hipdnnGetVersion() << "\n\n";

    std::cout << "Creating cuBLAS Handles!" << "\n";
    cublasErrorHandler( hipblasCreate(&cublasHandle) );
    int version;
    cublasGetVersion(cublasHandle,&version);
    std::cout << " Running cuBLAS version: " << version << "\n\n";
};

void cuNeuralNetworkbase::m_destroyHandles()
{
    std::cout << "\nDestroying cuDNN Handles!" << "\n";
    cudnnErrorHandler(hipdnnDestroyTensorDescriptor(dstTensorDesc));
    cudnnErrorHandler(hipdnnDestroyTensorDescriptor(srcTensorDesc));
    cudnnErrorHandler(hipdnnDestroy(cudnnHandle));

    std::cout << "Destroying cuBLAS Handles!" << "\n";
    cublasErrorHandler( hipblasDestroy(cublasHandle) );
};

/*void cuNeuralNetworkbase::fullyConnectedForward(
                           int& n, int& c, int& h, int& w,
                           float* srcData, float** dstData,
                           float* weight_d,float* bias_d)
{
    if (n != 1) {
        FatalError("Not Implemented");
    }
    int dim_x = c*h*w;
    int dim_y = ip.outputs;
    resize(dim_y, dstData);

    float alpha = float(1), beta = float(1);



    // place bias into dstData
    checkCudaErrors( hipMemcpy(*dstData, bias_d, dim_y*sizeof(float), hipMemcpyDeviceToDevice) );

    checkCudaErrors( hipblasSgemv(cublasHandle, HIPBLAS_OP_T,
                                 dim_x, dim_y,
                                 &alpha,
                                 weight_d, dim_x,
                                 srcData, 1,
                                 &beta,
                                 *dstData, 1) );

    h = 1;
    w = 1;
    c = dim_y;
}*/

/*void cuNeuralNetworkbase::activationForward(int n, int c, int h, int w, float* srcData, float** dstData)
{
    cudnnErrorHandler( hipdnnSetTensor4dDescriptor(srcTensorDesc,
                       HIPDNN_TENSOR_NCHW,
                       HIPDNN_DATA_FLOAT,
                       n, c,
                       h,
                       w) );
    cudnnErrorHandler( hipdnnSetTensor4dDescriptor(dstTensorDesc,
                       HIPDNN_TENSOR_NCHW,
                       HIPDNN_DATA_FLOAT,
                       n, c,
                       h,
                       w) );
    float alpha = 1.0f;
    float beta  = 0.0f;
    cudnnErrorHandler( hipdnnActivationForward(cudnnHandle,
                       HIPDNN_ACTIVATION_RELU,
                       &alpha,
                       srcTensorDesc,
                       srcData,
                       &beta,
                       dstTensorDesc,
                       *dstData) );
};*/
