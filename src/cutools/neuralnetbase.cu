// STD Lib Headers
#include <iostream>
#include <vector>
#include <sstream>
#include <string>

// CUDA Headers
#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include <hipblas.h>

#include "cudaerrorhandling.cuh"
#include "cudadevicestools.cuh"

#include "neuralnetbase.cuh"

void cuNeuralNetworkbase::setupCudaDevice()
{
    cudaErrorHandler(hipGetDeviceCount(&numdevice));
    devprops.resize(numdevice);

    /*GET CUDA DEVICE PROPERTIES*/
    for (int i=0; i<int(devprops.size()); ++i)
        cudaErrorHandler(hipGetDeviceProperties(&devprops[i],i));

    printDevProps(devprops);

    cudaErrorHandler(hipSetDevice(0));
};

void cuNeuralNetworkbase::createHandles()
{
    std::cout << "Running cuDNN version: " << hipdnnGetVersion() << "\n";
    std::cout << "Creating cuDNN Handles!" << "\n\n";
    cudnnErrorHandler(hipdnnCreate(&cudnnHandle));
    cudnnErrorHandler(hipdnnCreateTensorDescriptor(&srcTensorDesc));
    cudnnErrorHandler(hipdnnCreateTensorDescriptor(&dstTensorDesc));
};

void cuNeuralNetworkbase::destroyHandles()
{
    std::cout << "Destroying cuDNN Handles!" << "\n\n";
    cudnnErrorHandler(hipdnnDestroyTensorDescriptor(dstTensorDesc));
    cudnnErrorHandler(hipdnnDestroyTensorDescriptor(srcTensorDesc));
    cudnnErrorHandler(hipdnnDestroy(cudnnHandle));
};

void cuNeuralNetworkbase::fullyConnectedForward(const Layer_t& ip,
                           int& n, int& c, int& h, int& w,
                           value_type* srcData, value_type** dstData)
{
    if (n != 1) {
        FatalError("Not Implemented");
    }
    int dim_x = c*h*w;
    int dim_y = ip.outputs;
    resize(dim_y, dstData);

    value_type alpha = value_type(1), beta = value_type(1);
    // place bias into dstData
    checkCudaErrors( hipMemcpy(*dstData, ip.bias_d, dim_y*sizeof(value_type), hipMemcpyDeviceToDevice) );

    checkCudaErrors( hipblasSgemv(cublasHandle, HIPBLAS_OP_T,
                                 dim_x, dim_y,
                                 &alpha,
                                 ip.data_d, dim_x,
                                 srcData, 1,
                                 &beta,
                                 *dstData, 1) );

    h = 1;
    w = 1;
    c = dim_y;
}

void cuNeuralNetworkbase::activationForward(int n, int c, int h, int w, float* srcData, float** dstData)
{
    cudnnErrorHandler( hipdnnSetTensor4dDescriptor(srcTensorDesc,
                       HIPDNN_TENSOR_NCHW,
                       HIPDNN_DATA_FLOAT,
                       n, c,
                       h,
                       w) );
    cudnnErrorHandler( hipdnnSetTensor4dDescriptor(dstTensorDesc,
                       HIPDNN_TENSOR_NCHW,
                       HIPDNN_DATA_FLOAT,
                       n, c,
                       h,
                       w) );
    float alpha = 1.0f;
    float beta  = 0.0f;
    cudnnErrorHandler( hipdnnActivationForward(cudnnHandle,
                       HIPDNN_ACTIVATION_RELU,
                       &alpha,
                       srcTensorDesc,
                       srcData,
                       &beta,
                       dstTensorDesc,
                       *dstData) );
};
