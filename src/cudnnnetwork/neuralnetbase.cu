// STD Lib Headers
#include <iostream>
#include <vector>
#include <sstream>
#include <fstream>
#include <string>
#include <cstring>
#include <regex>
#include <signal.h>

// CUDA Headers
#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include <hipblas.h>

#include "../errorhandling.h"
#include "../tools/csvreader.hpp"

#include "../cutools/cudahosttools.cuh"
#include "../cutools/curandhosttools.cuh"

#include "neuralnetbase.cuh"

/*--------Setup CUDA Devices----------

Obtains number of devices, and device
properties.

--------------------------------------*/
void fpn::cuNeuralNetworkbase::m_setupCudaDevice() {
    cudaThrowHandler(hipGetDeviceCount(&numdevice));
    devprops.resize(numdevice);

    /*GET CUDA DEVICE PROPERTIES*/
    for (int i=0; i<int(devprops.size()); ++i)
        cudaThrowHandler(hipGetDeviceProperties(&devprops[i],i));

    printDevProps(devprops);

    cudaThrowHandler(hipSetDevice(0));
};

/*-------Setup CUDA Libraries---------

Creates cuDNN handles/descriptors and
cuBLAS handles for use by the child
classes.

--------------------------------------*/
void fpn::cuNeuralNetworkbase::m_createHandles() {
    std::cout << "Creating cuDNN Handle!" << "\n";
    cudnnThrowHandler(hipdnnCreate(&cudnnHandle));
    std::cout << " Running cuDNN version: " << hipdnnGetVersion() << "\n\n";
    std::cout << "Creating cuBLAS Handles!" << "\n";
    cublasThrowHandler( hipblasCreate(&cublasHandle) );
    int version;
    cublasGetVersion(cublasHandle,&version);
    std::cout << " Running cuBLAS version: " << version << "\n\n";
};

/*-------Destroy CUDA Libraries-------

Destroys cuDNN handles/descriptors and
cuBLAS handles that were defined in:

fpn::cuNeuralNetworkbase::m_createHandles()

--------------------------------------*/
void fpn::cuNeuralNetworkbase::m_destroyHandles() {
    std::cout << "Destroying cuDNN Handle!" << "\n";
    cudnnThrowHandler(hipdnnDestroy(cudnnHandle));
    std::cout << "Destroying cuBLAS Handles!" << "\n";
    cublasThrowHandler( hipblasDestroy(cublasHandle) );
};

/*-------Create Neural Network-------

--------------------------------------*/
void fpn::cuNeuralNetworkbase::m_createNetwork(const std::string templateString) {
    std::regex pattern_nntformat("^([0-9]{1,8}:){1,64}[0-9]{1,8}$"); // Ensure proper network template formatting
    if (!std::regex_search(templateString,pattern_nntformat)) {
        fpnThrowHandler(std::string("The network creation template syntax is incorrect."));
    }

    std::cout << "Creating a Neural Network from template " << std::endl;
    std::vector<unsigned int> netarch(m_parseNetworkTemplate(templateString));

    inlayersize = netarch.front();
    std::cout << " Neural net architecture requested: \n  Input layer size=" << netarch.front() << " followed by layers of size ";
    unsigned int Nw = 0;
    int Nb = 0;
    std::vector<unsigned int>::iterator it;
    for (it=netarch.begin()+1; it!=netarch.end(); ++it) {
        std::cout << *it << " ";
        Nw += *(it-1) **it;
        Nb += *it;
    }

    wbdataSize = ((Nw + Nb) * sizeof(float)) / float(1024*1024);
    if (trainer) {wbdataSize*=2.0;wbdataSize+=((Nb*sizeof(float))/float(1024*1024));} // Memory req doubled for training (derivatives)

    std::cout << "\n  Num. Weights: " << Nw << " -- Num. Biases: " << Nb << " required" << std::endl;
    std::cout << "  Network Device Memory Cost: " << wbdataSize << "MB" << std::endl;

    std::cout << "  Generating random weights and biases w/ cuRAND! " << std::endl;
    std::vector<float> rn;
    fpn::curandGenRandomFloats(rn,Nw+Nb);

    std::cout << "\n Building Neural Network Layers: " << std::endl;
    unsigned long long int idx=0;
    for (it=netarch.begin()+1; it!=netarch.end(); ++it) {
        Nw = *(it-1) **it;
        Nb = *it;
        std::cout << "   Layer " << it-netarch.begin()-1  << " w/ " << Nw << " weights and " << Nb << " bias.\n";

        std::vector<float> weight(Nw);
        std::vector<float> bias(Nb);

        std::memcpy(&weight[0],&rn[idx]   ,Nw*sizeof(float));
        std::memcpy(&bias[0]  ,&rn[idx+Nw],Nb*sizeof(float));

        idx += Nw+Nb;

        // Locally construct the class and emplace it on the layers vector
        layers.emplace_back(weight,bias,&cudnnHandle,&cublasHandle,trainer);
        // Load the data to the devices, must call clearDevice() to reset device data.
        layers.back().loadToDevice();
    }
};

/*-------Parse Network Template-------

Parse a string formatted as:
16:32:48:2
into a vector of uints. This is used
to define the neural network archit-
ecture.

--------------------------------------*/
std::vector<unsigned int> fpn::cuNeuralNetworkbase::m_parseNetworkTemplate(const std::string templateString) {
    std::vector<unsigned int> netarch;
    std::string wks(templateString);

    while (wks.find_first_of(":")!=std::string::npos) {
        size_t pos = wks.find_first_of(":");
        netarch.push_back(atoi(wks.substr(0,pos).c_str()));
        wks = wks.substr(pos+1);
    }

    netarch.push_back(atoi(wks.c_str()));

    return netarch;
};

/*---------Save Network Data----------

Load the network from the GPU and save
it to a file named in the argument
fname.

--------------------------------------*/
void fpn::cuNeuralNetworkbase::m_saveNetwork(const std::string &fname) {

    if (!layers.empty()) {
        std::cout << "\nSaving network data!" << std::endl;
        std::ofstream dataFile (fname);
        if (!dataFile) {
            std::stringstream _error;
            _error << "Error creating file: " << fname;
            fpnThrowHandler(_error.str());
        }

        for (auto l : layers) {
            l.loadFromDevice();
            dataFile << "$STARTLAYER\n";

            dataFile << "weights=";
            for (auto w : l.weightAccess())
                dataFile << w << ",";

            dataFile << "\n";

            dataFile << "biases=";
            for (auto b : l.biasAccess())
                dataFile << b << ",";

            dataFile << "\n";
        }

        dataFile.close();
    } else {
        std::cout << "\nCannot save data! Layers not loaded." << std::endl;
    }
};

/*---------Load Network Data----------

Load the network from the GPU and save
it to a file named in the argument
fname.

--------------------------------------*/
void fpn::cuNeuralNetworkbase::m_loadNetwork(const std::string &fname) {
    std::regex pattern_nnffile(".*\\.nnf$"); // Ensure only .nnf (Neural Network Format) files are given
    if (!std::regex_search(fname,pattern_nnffile)) {
        fpnThrowHandler(std::string("Only .nnf files can be used to construct the cuNeuralNetworkbase class"));
    }

    std::cout << "Loading the Neural Network data from file: " << fname << std::endl;

    std::string line;
    std::ifstream dataFile (fname.c_str());

    if (!dataFile) {
        std::stringstream _error;
        _error << "Error opening file: " << fname;
        fpnThrowHandler(_error.str());
    }

    int expline=-1;
    bool SAVE=false;
    if (dataFile.is_open()) {

        std::vector<float> weight_v,bias_v;
        while ( getline (dataFile,line) ) {
            if (expline==1) {
                std::string bias_s(line.substr(line.find_first_of("=")+1));
                csvreader(bias_s,bias_v);
                expline=-1;
                SAVE=true;
            }

            if (expline==0) {
                std::string weight_s(line.substr(line.find_first_of("=")+1));
                csvreader(weight_s,weight_v);
                expline=1;
            }

            if (line.find("$STARTLAYER")!=std::string::npos)
                expline=0;

            if (SAVE) {
                layers.emplace_back(weight_v,bias_v,&cudnnHandle,&cublasHandle,trainer);
                layers.back().loadToDevice();

                bias_v.clear();
                weight_v.clear();

                SAVE=false;
            }
        }
        dataFile.close();
    } else {
        std::cout << "NOT OPEN!" << std::endl;
    }
};
