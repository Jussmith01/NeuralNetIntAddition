// STD Lib Headers
#include <iostream>
#include <vector>
#include <sstream>
#include <string>
#include <regex>
#include <unistd.h>

// CUDA Headers
#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include <hipblas.h>

#include "../errorhandling.h"

#include "cudnnlayer_t.cuh"

// Declare Layer ID Counter
int fpn::ReLUlayer_t::IDindex = 0;

/*--------Resize a CUDA Container-------


--------------------------------------*/
void fpn::ReLUlayer_t::m_resize(int size, float **data) {
    if (*data != NULL) {
        cudaThrowHandler( hipFree(*data) );
    }
    cudaThrowHandler( hipMalloc(data, size*sizeof(float)) );
}

/*-----Load Layer Data to Device------


--------------------------------------*/
void fpn::ReLUlayer_t::m_loadDataToDevice() {
    int nw = weight_h.size();
    int nb =   bias_h.size();

    if (nw == 0 || nb == 0)
        fpnThrowHandler(std::string("Weights and/or biases cannot be empty."));

    cudnnThrowHandler(hipdnnCreateTensorDescriptor(&srcTensorDesc));
    cudnnThrowHandler(hipdnnCreateTensorDescriptor(&dstTensorDesc));

    /* Allocate Weights and Bias on Device */
    if (!dataLoad) {
        cudaThrowHandler(hipMalloc((void**)&weight_d,nw*sizeof(float)));
        cudaThrowHandler(hipMalloc((void**)&bias_d  ,nb*sizeof(float)));
    }

    /* Allocate Cost Derivatives on Device */
    if (trainer) {
        cudaThrowHandler(hipMalloc((void**)&dCdw_d,nw*sizeof(float)));
        cudaThrowHandler(hipMalloc((void**)&dCdb_d,nb*sizeof(float)));
        cudaThrowHandler(hipMalloc((void**)&Z_d,nb*sizeof(float)));
    }


    /* Copy Data */
    cudaThrowHandler(hipMemcpy(weight_d,&weight_h[0] ,nw*sizeof(float),hipMemcpyHostToDevice));
    cudaThrowHandler(hipMemcpy(bias_d  ,&bias_h[0]   ,nb*sizeof(float),hipMemcpyHostToDevice));

    dataLoad=true;
};

/*--------Clear Data on Device--------


--------------------------------------*/
void fpn::ReLUlayer_t::m_clearDataOnDevice() {
    weight_h.clear();
    bias_h.clear  ();

    cudaThrowHandler(hipDeviceSynchronize());

    cudnnThrowHandler(hipdnnDestroyTensorDescriptor(dstTensorDesc));
    cudnnThrowHandler(hipdnnDestroyTensorDescriptor(srcTensorDesc));

    if (dataLoad) {
        cudaThrowHandler(hipFree(weight_d));
        cudaThrowHandler(hipFree(bias_d  ));
    }

    if (trainer) {
        cudaThrowHandler(hipFree(dCdw_d));
        cudaThrowHandler(hipFree(dCdb_d));
        cudaThrowHandler(hipFree(Z_d));
    }

    dataLoad=false;
};

/*-----Retrieve Data from Device------


--------------------------------------*/
void fpn::ReLUlayer_t::m_retriveDataFromDevice() {
    int nw = weight_h.size();
    int nb =   bias_h.size();

    if (dataLoad) {
        cudaThrowHandler(hipMemcpy(&weight_h[0],weight_d,nw*sizeof(float),hipMemcpyDeviceToHost));
        cudaThrowHandler(hipMemcpy(&bias_h[0]  ,bias_d  ,nb*sizeof(float),hipMemcpyDeviceToHost));
    }
};

/*------Fully Connected Forward--------

Fully connected forward is called to
calculate the z values, which are used
to calculate the activations.

--------------------------------------*/
void fpn::ReLUlayer_t::fullyConnectedForward(int c,float* srcData, float** dstData) {
    if (n != 1) {
        fpnThrowHandler(std::string("Not Implemented"));
    }

    // c = data points
    // w = weights
    // b = biases
    // n = feature maps - NOT USED! Always 1;

    int dim_x = w*b;
    int dim_y = b;
    m_resize(dim_y, dstData);

    float alpha = float(1), beta = float(1);


    int ils = w/b; // Input layer size

    // place bias into dstData
    for (int i=0; i<c; ++i) {
        cudaThrowHandler( hipMemcpy(dstData[c*b], bias_d, dim_y*sizeof(float), hipMemcpyDeviceToDevice) );

        cublasThrowHandler( hipblasSgemv(*cublasHandle, HIPBLAS_OP_T,
                                        dim_x, dim_y,
                                        &alpha,
                                        weight_d, dim_x,
                                        &srcData[c*ils], 1,
                                        &beta,
                                        dstData[c*b], 1) );
    }
}

/*void cuNeuralNetworkbase::activationForward(int n, int c, int h, int w, float* srcData, float** dstData) {
    cudnnErrorHandler( hipdnnSetTensor4dDescriptor(srcTensorDesc,
                       HIPDNN_TENSOR_NCHW,
                       HIPDNN_DATA_FLOAT,
                       n, c,
                       h,
                       w) );
    cudnnErrorHandler( hipdnnSetTensor4dDescriptor(dstTensorDesc,
                       HIPDNN_TENSOR_NCHW,
                       HIPDNN_DATA_FLOAT,
                       n, c,
                       h,
                       w) );
    float alpha = 1.0f;
    float beta  = 0.0f;
    cudnnErrorHandler( hipdnnActivationForward(cudnnHandle,
                       HIPDNN_ACTIVATION_RELU,
                       &alpha,
                       srcTensorDesc,
                       srcData,
                       &beta,
                       dstTensorDesc,
                       *dstData) );
};*/
