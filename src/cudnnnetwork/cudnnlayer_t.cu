/*----------------------------------------------
        Written by Justin Smith ~August 2015
        E-Mail Jussmith48@gmail.com
        Copyright the Roitberg research group
        Chemistry Department
        University of Florida
        Gainesville FL.
------------------------------------------------*/
// STD Lib Headers
#include <iostream>
#include <vector>
#include <sstream>
#include <string>
#include <regex>
#include <unistd.h>

// CUDA Headers
#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include <hipblas.h>

#include "../errorhandling.h"

#include "../cutools/cudahosttools.cuh"
#include "cudnnlayer_t.cuh"

// Declare Layer ID Counter
int fpn::ReLUlayer_t::IDindex = 0;

/*-----Load Layer Data to Device------

Allocate device space and load data.

--------------------------------------*/
void fpn::ReLUlayer_t::m_loadDataToDevice() {
    w = weight_h.size();
    b =   bias_h.size();

    if ( w == 0 || b == 0 )
        fpnThrowHandler(std::string("Weights and/or biases cannot be empty."));

    cudnnThrowHandler(hipdnnCreateTensorDescriptor(&srcTensorDesc));
    cudnnThrowHandler(hipdnnCreateTensorDescriptor(&dstTensorDesc));

    /* Allocate Weights and Bias on Device */
    if (!dataLoad) {
        cudaThrowHandler(hipMalloc((void**)&weight_d,w*sizeof(float)));
        cudaThrowHandler(hipMalloc((void**)&bias_d  ,b*sizeof(float)));
    }

    /* Allocate Cost Derivatives and Z storage on Device, if training */
    if (trainer) {
        cudaThrowHandler(hipMalloc((void**)&dCdw_d,w*sizeof(float)));
        cudaThrowHandler(hipMalloc((void**)&dCdb_d,b*sizeof(float)));
        cudaThrowHandler(hipMalloc((void**)&Z_d,   b*sizeof(float)));
    }


    /* Copy Data */
    cudaThrowHandler(hipMemcpy(weight_d,&weight_h[0] ,w*sizeof(float),hipMemcpyHostToDevice));
    cudaThrowHandler(hipMemcpy(bias_d  ,&bias_h[0]   ,b*sizeof(float),hipMemcpyHostToDevice));

    dataLoad=true;
};

/*--------Clear Data on Device--------

Cleanup the device storage.

--------------------------------------*/
void fpn::ReLUlayer_t::m_clearDataOnDevice() {
    weight_h.clear();
    bias_h.clear  ();

    cudaThrowHandler(hipDeviceSynchronize());

    cudnnThrowHandler(hipdnnDestroyTensorDescriptor(dstTensorDesc));
    cudnnThrowHandler(hipdnnDestroyTensorDescriptor(srcTensorDesc));

    if (dataLoad) {
        cudaThrowHandler(hipFree(weight_d));
        cudaThrowHandler(hipFree(bias_d  ));
    }

    if (trainer) {
        cudaThrowHandler(hipFree(dCdw_d));
        cudaThrowHandler(hipFree(dCdb_d));
        cudaThrowHandler(hipFree(Z_d));
    }

    dataLoad=false;
};

/*-----Retrieve Data from Device------

Get weights and biases from the device.

--------------------------------------*/
void fpn::ReLUlayer_t::m_retriveDataFromDevice() {
    int nw = weight_h.size();
    int nb =   bias_h.size();

    if (dataLoad) {
        cudaThrowHandler(hipMemcpy(&weight_h[0],weight_d,nw*sizeof(float),hipMemcpyDeviceToHost));
        cudaThrowHandler(hipMemcpy(&bias_h[0]  ,bias_d  ,nb*sizeof(float),hipMemcpyDeviceToHost));
    }
};

/*------Fully Connected Forward--------

Fully connected forward is called to
calculate the z values, which are used
to calculate the activations.

--------------------------------------*/
void fpn::ReLUlayer_t::fullyConnectedForward(int c,float* srcData, float** dstData) {
    if (n != 1) {
        fpnThrowHandler(std::string("Not Implemented"));
    }

    // c = data points
    // w = weights
    // b = biases
    // n = feature maps - NOT USED! Always 1;

    int dim_x = w/b;
    int dim_y = b;
    cu_resize(dim_y*c,dstData);

    float alpha = float(1), beta = float(1);

    /* Copy Biases into the Dest set */
    cu_MemcpySmalltoLargeD2D(c,dim_y,bias_d,dstData);

    hipDeviceSynchronize();

    /* Feed forward all data via gemm */
    cublasThrowHandler( hipblasSgemm(*cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N,
                                    dim_y, c, dim_x,
                                    &alpha,
                                    weight_d, dim_x,
                                    srcData, dim_x,
                                    &beta,
                                    (*dstData),dim_y) );

    //printMatCudaData(dim_y,c,*dstData,"MULT dstData: ");
};

void fpn::ReLUlayer_t::activationForward(int c,float* srcData, float** dstData) {
    cu_resize(b*c,dstData);

    //printCudaData(b*c,srcData,"ACT1 srcData: ");

    cudnnThrowHandler( hipdnnSetTensor4dDescriptor(srcTensorDesc,
                       HIPDNN_TENSOR_NCHW,
                       HIPDNN_DATA_FLOAT,
                       1, n,
                       b,
                       c) );
    cudnnThrowHandler( hipdnnSetTensor4dDescriptor(dstTensorDesc,
                       HIPDNN_TENSOR_NCHW,
                       HIPDNN_DATA_FLOAT,
                       1, n,
                       b,
                       c) );
    float alpha = 1.0f;
    float beta  = 0.0f;
    cudnnThrowHandler( hipdnnActivationForward(*cudnnHandle,
                       HIPDNN_ACTIVATION_RELU,
                       &alpha,
                       srcTensorDesc,
                       srcData,
                       &beta,
                       dstTensorDesc,
                       *dstData) );

    //printCudaData(b*c,dstData,"ACT2 dstData: ");
};
