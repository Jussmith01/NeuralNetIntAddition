// STD Lib Headers
#include <iostream>
#include <vector>
#include <sstream>
#include <string>
#include <regex>
#include <unistd.h>

// CUDA Headers
#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include <hipblas.h>

#include "../errorhandling.h"

#include "cudnnlayer_t.cuh"

// Declare Layer ID Counter
int fpn::ReLUlayer_t::IDindex = 0;

/*-----Load Layer Data to Device------


--------------------------------------*/
void fpn::ReLUlayer_t::m_loadDataToDevice() {
    int nw = weight_h.size();
    int nb =   bias_h.size();

    if (nw == 0 || nb == 0)
        fpnThrowHandler(std::string("Weights and/or biases cannot be empty."));

    cudnnThrowHandler(hipdnnCreateTensorDescriptor(&srcTensorDesc));
    cudnnThrowHandler(hipdnnCreateTensorDescriptor(&dstTensorDesc));

    /* Allocate Weights and Bias on Device */
    if (!dataLoad)
    {
        cudaThrowHandler(hipMalloc((void**)&weight_d,nw*sizeof(float)));
        cudaThrowHandler(hipMalloc((void**)&bias_d  ,nb*sizeof(float)));
    }

    /* Allocate Cost Derivatives on Device */
    if (trainer)
    {
        cudaThrowHandler(hipMalloc((void**)&dCdw_d,nw*sizeof(float)));
        cudaThrowHandler(hipMalloc((void**)&dCdb_d,nb*sizeof(float)));
        cudaThrowHandler(hipMalloc((void**)&Z_d,nb*sizeof(float)));
    }


    /* Copy Data */
    cudaThrowHandler(hipMemcpy(weight_d,&weight_h[0] ,nw*sizeof(float),hipMemcpyHostToDevice));
    cudaThrowHandler(hipMemcpy(bias_d  ,&bias_h[0]   ,nb*sizeof(float),hipMemcpyHostToDevice));

    dataLoad=true;
};

/*--------Clear Data on Device--------


--------------------------------------*/
void fpn::ReLUlayer_t::m_clearDataOnDevice() {
     weight_h.clear();
     bias_h.clear  ();

     cudaThrowHandler(hipDeviceSynchronize());

     cudnnThrowHandler(hipdnnDestroyTensorDescriptor(dstTensorDesc));
     cudnnThrowHandler(hipdnnDestroyTensorDescriptor(srcTensorDesc));

     if (dataLoad)
     {
        cudaThrowHandler(hipFree(weight_d));
        cudaThrowHandler(hipFree(bias_d  ));
     }

    if (trainer)
    {
        cudaThrowHandler(hipFree(dCdw_d));
        cudaThrowHandler(hipFree(dCdb_d));
        cudaThrowHandler(hipFree(Z_d));
    }

     dataLoad=false;
};

/*-----Retrieve Data from Device------


--------------------------------------*/
void fpn::ReLUlayer_t::m_retriveDataFromDevice() {
    int nw = weight_h.size();
    int nb =   bias_h.size();

    if (dataLoad)
    {
        cudaThrowHandler(hipMemcpy(&weight_h[0],weight_d,nw*sizeof(float),hipMemcpyDeviceToHost));
        cudaThrowHandler(hipMemcpy(&bias_h[0]  ,bias_d  ,nb*sizeof(float),hipMemcpyDeviceToHost));
    }
};

/*------Fully Connected Forward--------

Fully connected forward is called to
calculate the z values, which are used
to calculate the activations.

--------------------------------------*/
void fpn::ReLUlayer_t::fullyConnectedForward(float* srcData, float** dstData) {
    if (n != 1) {
        fpnThrowHandler(std::string("Not Implemented"));
    }

    int dim_x = c*h*w;
    int dim_y = n;
    //resize(dim_y, dstData);

    float alpha = float(1), beta = float(1);



    // place bias into dstData
    cudaThrowHandler( hipMemcpy(*dstData, bias_d, dim_y*sizeof(float), hipMemcpyDeviceToDevice) );

    cublasThrowHandler( hipblasSgemv(*cublasHandle, HIPBLAS_OP_T,
                                 dim_x, dim_y,
                                 &alpha,
                                 weight_d, dim_x,
                                 srcData, 1,
                                 &beta,
                                 *dstData, 1) );

    h = 1;
    w = 1;
    c = dim_y;
}

/*void cuNeuralNetworkbase::activationForward(int n, int c, int h, int w, float* srcData, float** dstData) {
    cudnnErrorHandler( hipdnnSetTensor4dDescriptor(srcTensorDesc,
                       HIPDNN_TENSOR_NCHW,
                       HIPDNN_DATA_FLOAT,
                       n, c,
                       h,
                       w) );
    cudnnErrorHandler( hipdnnSetTensor4dDescriptor(dstTensorDesc,
                       HIPDNN_TENSOR_NCHW,
                       HIPDNN_DATA_FLOAT,
                       n, c,
                       h,
                       w) );
    float alpha = 1.0f;
    float beta  = 0.0f;
    cudnnErrorHandler( hipdnnActivationForward(cudnnHandle,
                       HIPDNN_ACTIVATION_RELU,
                       &alpha,
                       srcTensorDesc,
                       srcData,
                       &beta,
                       dstTensorDesc,
                       *dstData) );
};*/
