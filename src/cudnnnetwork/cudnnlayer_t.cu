// STD Lib Headers
#include <iostream>
#include <vector>
#include <sstream>
#include <string>
#include <regex>
#include <unistd.h>

// CUDA Headers
#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include <hipblas.h>

#include "../errorhandling.h"

#include "cudnnlayer_t.cuh"

// Declare Layer ID Counter
int fpn::ReLUlayer_t::IDindex = 0;

/*-----Load Layer Data to Device------


--------------------------------------*/
void fpn::ReLUlayer_t::m_loadDataToDevice() {
    int nw = weight_h.size();
    int nb =   bias_h.size();

    if (nw == 0 || nb == 0)
        fpnThrowHandler(std::string("Weights and/or biases cannot be empty."));

    /* Allocate Device */
    if (!dataLoad)
    {
        cudaThrowHandler(hipMalloc((void**)&weight_d,nw*sizeof(float)));
        cudaThrowHandler(hipMalloc((void**)&bias_d  ,nb*sizeof(float)));
    }

    /* Copy Data */
    cudaThrowHandler(hipMemcpy(weight_d,&weight_h[0] ,nw*sizeof(float),hipMemcpyHostToDevice));
    cudaThrowHandler(hipMemcpy(bias_d  ,&bias_h[0]   ,nb*sizeof(float),hipMemcpyHostToDevice));

    dataLoad=true;
};

/*--------Clear Data on Device--------


--------------------------------------*/
void fpn::ReLUlayer_t::m_clearDataOnDevice() {
     weight_h.clear();
     bias_h.clear  ();

     cudaThrowHandler(hipDeviceSynchronize());

     if (dataLoad)
     {
        cudaThrowHandler(hipFree(weight_d));
        cudaThrowHandler(hipFree(bias_d  ));
     }

     dataLoad=false;
};

/*-----Retrieve Data from Device------


--------------------------------------*/
void fpn::ReLUlayer_t::m_retriveDataFromDevice() {
    int nw = weight_h.size();
    int nb =   bias_h.size();

    if (dataLoad)
    {
        cudaThrowHandler(hipMemcpy(&weight_h[0],weight_d,nw*sizeof(float),hipMemcpyDeviceToHost));
        cudaThrowHandler(hipMemcpy(&bias_h[0]  ,bias_d  ,nb*sizeof(float),hipMemcpyDeviceToHost));
    }
};

