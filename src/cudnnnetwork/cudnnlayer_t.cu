// STD Lib Headers
#include <iostream>
#include <vector>
#include <sstream>
#include <string>
#include <regex>
#include <unistd.h>

// CUDA Headers
#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include <hipblas.h>

#include "../errorhandling.h"

#include "cudnnlayer_t.cuh"

// Declare Layer ID Counter
int fpn::ReLUlayer_t::IDindex = 0;

void printCudaData(int size,float *data)
{
    std::vector<float> test(size);
    cudaThrowHandler( hipMemcpy(&test[0],data,size*sizeof(float),hipMemcpyDeviceToHost) );
    std::cout << "CHECK DATA:\n";
    for (auto i : test)
    {
        std::cout << i << " ";
    }
    std::cout << "\n";
};


/*--------Resize a CUDA Container-------


--------------------------------------*/
void fpn::ReLUlayer_t::m_resize(int size, float *data) {
    if (data != NULL) {
        cudaThrowHandler( hipFree(data) );
    }
    cudaThrowHandler( hipMalloc((void**)&data, size*sizeof(float)) );
}

/*-----Load Layer Data to Device------


--------------------------------------*/
void fpn::ReLUlayer_t::m_loadDataToDevice() {
    w = weight_h.size();
    b =   bias_h.size();

    if (w == 0 || b == 0)
        fpnThrowHandler(std::string("Weights and/or biases cannot be empty."));

    cudnnThrowHandler(hipdnnCreateTensorDescriptor(&srcTensorDesc));
    cudnnThrowHandler(hipdnnCreateTensorDescriptor(&dstTensorDesc));

    /* Allocate Weights and Bias on Device */
    if (!dataLoad) {
        cudaThrowHandler(hipMalloc((void**)&weight_d,w*sizeof(float)));
        cudaThrowHandler(hipMalloc((void**)&bias_d  ,b*sizeof(float)));
    }

    /* Allocate Cost Derivatives on Device */
    if (trainer) {
        cudaThrowHandler(hipMalloc((void**)&dCdw_d,w*sizeof(float)));
        cudaThrowHandler(hipMalloc((void**)&dCdb_d,b*sizeof(float)));
        cudaThrowHandler(hipMalloc((void**)&Z_d,b*sizeof(float)));
    }


    /* Copy Data */
    cudaThrowHandler(hipMemcpy(weight_d,&weight_h[0] ,w*sizeof(float),hipMemcpyHostToDevice));
    cudaThrowHandler(hipMemcpy(bias_d  ,&bias_h[0]   ,b*sizeof(float),hipMemcpyHostToDevice));

    dataLoad=true;
};

/*--------Clear Data on Device--------


--------------------------------------*/
void fpn::ReLUlayer_t::m_clearDataOnDevice() {
    weight_h.clear();
    bias_h.clear  ();

    cudaThrowHandler(hipDeviceSynchronize());

    cudnnThrowHandler(hipdnnDestroyTensorDescriptor(dstTensorDesc));
    cudnnThrowHandler(hipdnnDestroyTensorDescriptor(srcTensorDesc));

    if (dataLoad) {
        cudaThrowHandler(hipFree(weight_d));
        cudaThrowHandler(hipFree(bias_d  ));
    }

    if (trainer) {
        cudaThrowHandler(hipFree(dCdw_d));
        cudaThrowHandler(hipFree(dCdb_d));
        cudaThrowHandler(hipFree(Z_d));
    }

    dataLoad=false;
};

/*-----Retrieve Data from Device------


--------------------------------------*/
void fpn::ReLUlayer_t::m_retriveDataFromDevice() {
    int nw = weight_h.size();
    int nb =   bias_h.size();

    if (dataLoad) {
        cudaThrowHandler(hipMemcpy(&weight_h[0],weight_d,nw*sizeof(float),hipMemcpyDeviceToHost));
        cudaThrowHandler(hipMemcpy(&bias_h[0]  ,bias_d  ,nb*sizeof(float),hipMemcpyDeviceToHost));
    }
};

/*------Fully Connected Forward--------

Fully connected forward is called to
calculate the z values, which are used
to calculate the activations.

--------------------------------------*/
void fpn::ReLUlayer_t::fullyConnectedForward(int c,float* srcData, float* dstData) {
    if (n != 1) {
        fpnThrowHandler(std::string("Not Implemented"));
    }

    // c = data points
    // w = weights
    // b = biases
    // n = feature maps - NOT USED! Always 1;

    int dim_x = w/b;
    int dim_y = b;
    m_resize(dim_y*c,dstData);

    float alpha = float(1), beta = float(1);

    // place bias into dstData
    for (int i=0; i<c; ++i) {
        cudaThrowHandler( hipMemcpy(dstData+i*dim_y,bias_d,dim_y*sizeof(float),hipMemcpyDeviceToDevice) );
        cublasThrowHandler( hipblasSgemv(*cublasHandle, HIPBLAS_OP_T,
                                        dim_x, dim_y,
                                        &alpha,
                                        weight_d, dim_x,
                                        srcData, 1,
                                        &beta,
                                        dstData, 1) );
    }

    //printCudaData(dim_y*c,dstData);
};

/*void cuNeuralNetworkbase::activationForward(int n, int c, int h, int w, float* srcData, float** dstData) {
    cudnnErrorHandler( hipdnnSetTensor4dDescriptor(srcTensorDesc,
                       HIPDNN_TENSOR_NCHW,
                       HIPDNN_DATA_FLOAT,
                       n, c,
                       h,
                       w) );
    cudnnErrorHandler( hipdnnSetTensor4dDescriptor(dstTensorDesc,
                       HIPDNN_TENSOR_NCHW,
                       HIPDNN_DATA_FLOAT,
                       n, c,
                       h,
                       w) );
    float alpha = 1.0f;
    float beta  = 0.0f;
    cudnnErrorHandler( hipdnnActivationForward(cudnnHandle,
                       HIPDNN_ACTIVATION_RELU,
                       &alpha,
                       srcTensorDesc,
                       srcData,
                       &beta,
                       dstTensorDesc,
                       *dstData) );
};*/
